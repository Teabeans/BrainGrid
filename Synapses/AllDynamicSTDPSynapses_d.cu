/*
 * AllDynamicSTDPSynapses_d.cu
 *
 */

#include "AllDynamicSTDPSynapses.h"
#include "AllSynapsesDeviceFuncs.h"
#include "Book.h"

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesDevice  Reference to the AllDynamicSTDPSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllDynamicSTDPSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice, const SimulationInfo *sim_info ) {
	allocSynapseDeviceStruct( allSynapsesDevice, sim_info->totalNeurons, sim_info->maxSynapsesPerNeuron );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesDevice     Reference to the AllDynamicSTDPSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllDynamicSTDPSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice, int num_neurons, int maxSynapsesPerNeuron ) {
	AllDynamicSTDPSynapsesDeviceProperties allSynapses;

	allocDeviceStruct( allSynapses, num_neurons, maxSynapsesPerNeuron );

	HANDLE_ERROR( hipMalloc( allSynapsesDevice, sizeof( AllDynamicSTDPSynapsesDeviceProperties ) ) );
	HANDLE_ERROR( hipMemcpy ( *allSynapsesDevice, &allSynapses, sizeof( AllDynamicSTDPSynapsesDeviceProperties ), hipMemcpyHostToDevice ) );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *  (Helper function of allocSynapseDeviceStruct)
 *
 *  @param  allSynapsesDevice     Reference to the AllDynamicSTDPSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllDynamicSTDPSynapses::allocDeviceStruct( AllDynamicSTDPSynapsesDeviceProperties &allSynapses, int num_neurons, int maxSynapsesPerNeuron ) {
        AllSTDPSynapses::allocDeviceStruct( allSynapses, num_neurons, maxSynapsesPerNeuron );

        BGSIZE max_total_synapses = maxSynapsesPerNeuron * num_neurons;

        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.lastSpike, max_total_synapses * sizeof( uint64_t ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.r, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.u, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.D, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.U, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.F, max_total_synapses * sizeof( BGFLOAT ) ) );
}

/*
 *  Delete GPU memories.
 *
 *  @param  allSynapsesDevice  Reference to the AllDynamicSTDPSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllDynamicSTDPSynapses::deleteSynapseDeviceStruct( void* allSynapsesDevice ) {
	AllDynamicSTDPSynapsesDeviceProperties allSynapses;

	HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllDynamicSTDPSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

	deleteDeviceStruct( allSynapses );

	HANDLE_ERROR( hipFree( allSynapsesDevice ) );
}

/*
 *  Delete GPU memories.
 *  (Helper function of deleteSynapseDeviceStruct)
 *
 *  @param  allSynapsesDevice  Reference to the AllDynamicSTDPSynapsesDeviceProperties struct 
 *                             on device memory.
 */
void AllDynamicSTDPSynapses::deleteDeviceStruct( AllDynamicSTDPSynapsesDeviceProperties& allSynapses ) {
        HANDLE_ERROR( hipFree( allSynapses.lastSpike ) );
	HANDLE_ERROR( hipFree( allSynapses.r ) );
	HANDLE_ERROR( hipFree( allSynapses.u ) );
	HANDLE_ERROR( hipFree( allSynapses.D ) );
	HANDLE_ERROR( hipFree( allSynapses.U ) );
	HANDLE_ERROR( hipFree( allSynapses.F ) );

        AllSTDPSynapses::deleteDeviceStruct( allSynapses );
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesDevice  Reference to the AllDynamicSTDPSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllDynamicSTDPSynapses::copySynapseHostToDevice( void* allSynapsesDevice, const SimulationInfo *sim_info ) { // copy everything necessary
	copySynapseHostToDevice( allSynapsesDevice, sim_info->totalNeurons, sim_info->maxSynapsesPerNeuron );	
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesDevice     Reference to the AllDynamicSTDPSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllDynamicSTDPSynapses::copySynapseHostToDevice( void* allSynapsesDevice, int num_neurons, int maxSynapsesPerNeuron ) { // copy everything necessary
	AllDynamicSTDPSynapsesDeviceProperties allSynapses;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllDynamicSTDPSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

	copyHostToDevice( allSynapsesDevice, allSynapses, num_neurons, maxSynapsesPerNeuron );	
}

/*
 *  Copy all synapses' data from host to device.
 *  (Helper function of copySynapseHostToDevice)
 *
 *  @param  allSynapsesDevice     Reference to the AllDynamicSTDPSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllDynamicSTDPSynapses::copyHostToDevice( void* allSynapsesDevice, AllDynamicSTDPSynapsesDeviceProperties& allSynapses, int num_neurons, int maxSynapsesPerNeuron ) { // copy everything necessary 
        AllSTDPSynapses::copyHostToDevice( allSynapsesDevice, allSynapses, num_neurons, maxSynapsesPerNeuron );

        BGSIZE max_total_synapses = maxSynapsesPerNeuron * num_neurons;
        
        HANDLE_ERROR( hipMemcpy ( allSynapses.lastSpike, lastSpike,
                max_total_synapses * sizeof( uint64_t ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.r, r,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.u, u,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.D, D,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.U, U,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.F, F,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
}

/*
 *  Copy all synapses' data from device to host.
 *
 *  @param  allSynapsesDevice  Reference to the AllDynamicSTDPSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllDynamicSTDPSynapses::copySynapseDeviceToHost( void* allSynapsesDevice, const SimulationInfo *sim_info ) {
	// copy everything necessary
	AllDynamicSTDPSynapsesDeviceProperties allSynapses;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllDynamicSTDPSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

	copyDeviceToHost( allSynapses, sim_info );
}

/*
 *  Copy all synapses' data from device to host.
 *  (Helper function of copySynapseDeviceToHost)
 *
 *  @param  allSynapsesDevice     Reference to the AllDynamicSTDPSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllDynamicSTDPSynapses::copyDeviceToHost( AllDynamicSTDPSynapsesDeviceProperties& allSynapses, const SimulationInfo *sim_info ) {
        AllSTDPSynapses::copyDeviceToHost( allSynapses, sim_info ) ;

	int num_neurons = sim_info->totalNeurons;
	BGSIZE max_total_synapses = sim_info->maxSynapsesPerNeuron * num_neurons;

        HANDLE_ERROR( hipMemcpy ( lastSpike, allSynapses.lastSpike,
                max_total_synapses * sizeof( uint64_t ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( r, allSynapses.r,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( u, allSynapses.u,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( D, allSynapses.D,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( U, allSynapses.U,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( F, allSynapses.F,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
}

/**     
 *  Set synapse class ID defined by enumClassSynapses for the caller's Synapse class.
 *  The class ID will be set to classSynapses_d in device memory,
 *  and the classSynapses_d will be referred to call a device function for the
 *  particular synapse class.
 *  Because we cannot use virtual function (Polymorphism) in device functions,
 *  we use this scheme.
 *  Note: we used to use a function pointer; however, it caused the growth_cuda crash
 *  (see issue#137).
 */
void AllDynamicSTDPSynapses::setSynapseClassID()
{
    enumClassSynapses classSynapses_h = classAllDynamicSTDPSynapses;

    HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(classSynapses_d), &classSynapses_h, sizeof(enumClassSynapses)) );
}

/*
 *  Prints GPU SynapsesProps data.
 *   
 *  @param  allSynapsesDeviceProps   Reference to the corresponding SynapsesDeviceProperties struct on device memory.
 */
void AllDynamicSTDPSynapses::printGPUSynapsesProps( void* allSynapsesDeviceProps ) const
{
    AllDynamicSTDPSynapsesDeviceProperties allSynapsesProps;

    //allocate print out data members
    BGSIZE size = maxSynapsesPerNeuron * count_neurons;
    if (size != 0) {
        BGSIZE *synapse_countsPrint = new BGSIZE[count_neurons];
        BGSIZE maxSynapsesPerNeuronPrint;
        BGSIZE total_synapse_countsPrint;
        int count_neuronsPrint;
        int *sourceNeuronIndexPrint = new int[size];
        int *destNeuronIndexPrint = new int[size];
        BGFLOAT *WPrint = new BGFLOAT[size];

        synapseType *typePrint = new synapseType[size];
        BGFLOAT *psrPrint = new BGFLOAT[size];
        bool *in_usePrint = new bool[size];

        for (BGSIZE i = 0; i < size; i++) {
            in_usePrint[i] = false;
        }

        for (int i = 0; i < count_neurons; i++) {
            synapse_countsPrint[i] = 0;
        }

        BGFLOAT *decayPrint = new BGFLOAT[size];
        int *total_delayPrint = new int[size];
        BGFLOAT *tauPrint = new BGFLOAT[size];

        int *total_delayPostPrint = new int[size];
        BGFLOAT *tauspostPrint = new BGFLOAT[size];
        BGFLOAT *tausprePrint = new BGFLOAT[size];
        BGFLOAT *tauposPrint = new BGFLOAT[size];
        BGFLOAT *taunegPrint = new BGFLOAT[size];
        BGFLOAT *STDPgapPrint = new BGFLOAT[size];
        BGFLOAT *WexPrint = new BGFLOAT[size];
        BGFLOAT *AnegPrint = new BGFLOAT[size];
        BGFLOAT *AposPrint = new BGFLOAT[size];
        BGFLOAT *muposPrint = new BGFLOAT[size];
        BGFLOAT *munegPrint = new BGFLOAT[size];
        bool *useFroemkeDanSTDPPrint = new bool[size];

        uint64_t *lastSpikePrint = new uint64_t[size];
        BGFLOAT *rPrint = new BGFLOAT[size];
        BGFLOAT *uPrint = new BGFLOAT[size];
        BGFLOAT *DPrint = new BGFLOAT[size];
        BGFLOAT *UPrint = new BGFLOAT[size];
        BGFLOAT *FPrint = new BGFLOAT[size];

        // copy everything
        HANDLE_ERROR( hipMemcpy ( &allSynapsesProps, allSynapsesDeviceProps, sizeof( AllDynamicSTDPSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( synapse_countsPrint, allSynapsesProps.synapse_counts, count_neurons * sizeof( BGSIZE ), hipMemcpyDeviceToHost ) );
        maxSynapsesPerNeuronPrint = allSynapsesProps.maxSynapsesPerNeuron;
        total_synapse_countsPrint = allSynapsesProps.total_synapse_counts;
        count_neuronsPrint = allSynapsesProps.count_neurons;

        // Set count_neurons to 0 to avoid illegal memory deallocation
        // at AllSynapsesProps deconstructor.
        allSynapsesProps.count_neurons = 0;

        HANDLE_ERROR( hipMemcpy ( sourceNeuronIndexPrint, allSynapsesProps.sourceNeuronIndex, size * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( destNeuronIndexPrint, allSynapsesProps.destNeuronIndex, size * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( WPrint, allSynapsesProps.W, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( typePrint, allSynapsesProps.type, size * sizeof( synapseType ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( psrPrint, allSynapsesProps.psr, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( in_usePrint, allSynapsesProps.in_use, size * sizeof( bool ), hipMemcpyDeviceToHost ) );

        HANDLE_ERROR( hipMemcpy ( decayPrint, allSynapsesProps.decay, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tauPrint, allSynapsesProps.tau, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( total_delayPrint, allSynapsesProps.total_delay,size * sizeof( int ), hipMemcpyDeviceToHost ) );

        HANDLE_ERROR( hipMemcpy ( total_delayPostPrint, allSynapsesProps.total_delayPost, size * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tauspostPrint, allSynapsesProps.tauspost, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tausprePrint, allSynapsesProps.tauspre, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tauposPrint, allSynapsesProps.taupos, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( taunegPrint, allSynapsesProps.tauneg, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( STDPgapPrint, allSynapsesProps.STDPgap, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( WexPrint, allSynapsesProps.Wex, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( AnegPrint, allSynapsesProps.Aneg, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( AposPrint, allSynapsesProps.Apos, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( muposPrint, allSynapsesProps.mupos, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( munegPrint, allSynapsesProps.muneg, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( useFroemkeDanSTDPPrint, allSynapsesProps.useFroemkeDanSTDP, size * sizeof( bool ), hipMemcpyDeviceToHost ) );

        HANDLE_ERROR( hipMemcpy ( lastSpikePrint, allSynapsesProps.lastSpike, size * sizeof( uint64_t ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( rPrint, allSynapsesProps.r, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( uPrint, allSynapsesProps.u, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( DPrint, allSynapsesProps.D, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( UPrint, allSynapsesProps.U, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( FPrint, allSynapsesProps.F, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );

        for(int i = 0; i < maxSynapsesPerNeuron * count_neurons; i++) {
            if (WPrint[i] != 0.0) {
                cout << "GPU W[" << i << "] = " << WPrint[i];
                cout << " GPU sourNeuron: " << sourceNeuronIndexPrint[i];
                cout << " GPU desNeuron: " << destNeuronIndexPrint[i];
                cout << " GPU type: " << typePrint[i];
                cout << " GPU psr: " << psrPrint[i];
                cout << " GPU in_use:" << in_usePrint[i];

                cout << " GPU decay: " << decayPrint[i];
                cout << " GPU tau: " << tauPrint[i];
                cout << " GPU total_delay: " << total_delayPrint[i];

                cout << " GPU total_delayPost: " << total_delayPostPrint[i];
                cout << " GPU tauspost: " << tauspostPrint[i];
                cout << " GPU tauspre: " << tausprePrint[i];
                cout << " GPU taupos: " << tauposPrint[i];
                cout << " GPU tauneg: " << taunegPrint[i];
                cout << " GPU STDPgap: " << STDPgapPrint[i];
                cout << " GPU Wex: " << WexPrint[i];
                cout << " GPU Aneg: " << AnegPrint[i];
                cout << " GPU Apos: " << AposPrint[i];
                cout << " GPU mupos: " << muposPrint[i];
                cout << " GPU muneg: " << munegPrint[i];
                cout << " GPU useFroemkeDanSTDP: " << useFroemkeDanSTDPPrint[i];

                cout << " GPU lastSpike: " << lastSpikePrint[i];
                cout << " GPU r: " << rPrint[i];
                cout << " GPU u: " << uPrint[i];
                cout << " GPU D: " << DPrint[i];
                cout << " GPU U: " << UPrint[i];
                cout << " GPU F: " << FPrint[i] << endl;
            }
        }

        for (int i = 0; i < count_neurons; i++) {
            cout << "GPU synapse_counts:" << "neuron[" << i  << "]" << synapse_countsPrint[i] << endl;
        }

        cout << "GPU total_synapse_counts:" << total_synapse_countsPrint << endl;
        cout << "GPU maxSynapsesPerNeuron:" << maxSynapsesPerNeuronPrint << endl;
        cout << "GPU count_neurons:" << count_neuronsPrint << endl;


        // Set count_neurons to 0 to avoid illegal memory deallocation
        // at AllDSSynapsesProps deconstructor.
        allSynapsesProps.count_neurons = 0;

        delete[] destNeuronIndexPrint;
        delete[] WPrint;
        delete[] sourceNeuronIndexPrint;
        delete[] psrPrint;
        delete[] typePrint;
        delete[] in_usePrint;
        delete[] synapse_countsPrint;
        destNeuronIndexPrint = NULL;
        WPrint = NULL;
        sourceNeuronIndexPrint = NULL;
        psrPrint = NULL;
        typePrint = NULL;
        in_usePrint = NULL;
        synapse_countsPrint = NULL;

        delete[] decayPrint;
        delete[] total_delayPrint;
        delete[] tauPrint;
        decayPrint = NULL;
        total_delayPrint = NULL;
        tauPrint = NULL;

        delete[] total_delayPostPrint;
        delete[] tauspostPrint;
        delete[] tausprePrint;
        delete[] tauposPrint;
        delete[] taunegPrint;
        delete[] STDPgapPrint;
        delete[] WexPrint;
        delete[] AnegPrint;
        delete[] AposPrint;
        delete[] muposPrint;
        delete[] munegPrint;
        delete[] useFroemkeDanSTDPPrint;
        total_delayPostPrint = NULL;
        tauspostPrint = NULL;
        tausprePrint = NULL;
        tauposPrint = NULL;
        taunegPrint = NULL;
        STDPgapPrint = NULL;
        WexPrint = NULL;
        AnegPrint = NULL;
        AposPrint = NULL;
        muposPrint = NULL;
        munegPrint = NULL;
        useFroemkeDanSTDPPrint = NULL;

        delete[] lastSpikePrint;
        delete[] rPrint;
        delete[] uPrint;
        delete[] DPrint;
        delete[] UPrint;
        delete[] FPrint;
        lastSpikePrint = NULL;
        rPrint = NULL;
        uPrint = NULL;
        DPrint = NULL;
        UPrint = NULL;
        FPrint = NULL;
    }
}

