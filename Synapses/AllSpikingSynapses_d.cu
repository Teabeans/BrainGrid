/*
 * AllSpikingSynapses.cu
 *
 */

#include "AllSpikingSynapses.h"
#include "AllSynapsesDeviceFuncs.h"
#include <hip/hip_runtime_api.h>

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesDevice  Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllSpikingSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice, const SimulationInfo *sim_info, const ClusterInfo *clr_info ) {
        allocSynapseDeviceStruct( allSynapsesDevice, clr_info->totalClusterNeurons, sim_info->maxSynapsesPerNeuron, clr_info->clusterID );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesDevice     Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 *  @param  clusterID             The cluster ID of the cluster.
 */
void AllSpikingSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice, int num_neurons, int maxSynapsesPerNeuron, CLUSTER_INDEX_TYPE clusterID ) {
        AllSpikingSynapsesDeviceProperties allSynapses;

        allocDeviceStruct( allSynapses, num_neurons, maxSynapsesPerNeuron, clusterID );

        checkCudaErrors( hipMalloc( allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ) ) );
        checkCudaErrors( hipMemcpy ( *allSynapsesDevice, &allSynapses, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyHostToDevice ) );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *  (Helper function of allocSynapseDeviceStruct)
 *
 *  @param  allSynapsesDevice     Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 *  @param  clusterID             The cluster ID of the cluster.
 */
void AllSpikingSynapses::allocDeviceStruct( AllSpikingSynapsesDeviceProperties &allSynapses, int num_neurons, int maxSynapsesPerNeuron, CLUSTER_INDEX_TYPE clusterID ) {
        BGSIZE max_total_synapses = maxSynapsesPerNeuron * num_neurons;

        checkCudaErrors( hipMalloc( ( void ** ) &allSynapses.sourceNeuronLayoutIndex, max_total_synapses * sizeof( int ) ) );
        checkCudaErrors( hipMalloc( ( void ** ) &allSynapses.destNeuronLayoutIndex, max_total_synapses * sizeof( int ) ) );
        checkCudaErrors( hipMalloc( ( void ** ) &allSynapses.W, max_total_synapses * sizeof( BGFLOAT ) ) );
        checkCudaErrors( hipMalloc( ( void ** ) &allSynapses.type, max_total_synapses * sizeof( synapseType ) ) );
        checkCudaErrors( hipMalloc( ( void ** ) &allSynapses.psr, max_total_synapses * sizeof( BGFLOAT ) ) );
        checkCudaErrors( hipMalloc( ( void ** ) &allSynapses.in_use, max_total_synapses * sizeof( bool ) ) );
        checkCudaErrors( hipMalloc( ( void ** ) &allSynapses.synapse_counts, num_neurons * sizeof( BGSIZE ) ) );
        checkCudaErrors( hipMalloc( ( void ** ) &allSynapses.decay, max_total_synapses * sizeof( BGFLOAT ) ) );
        checkCudaErrors( hipMalloc( ( void ** ) &allSynapses.tau, max_total_synapses * sizeof( BGFLOAT ) ) );
        checkCudaErrors( hipMalloc( ( void ** ) &allSynapses.total_delay, max_total_synapses * sizeof( int ) ) );
        checkCudaErrors( hipMalloc( ( void ** ) &allSynapses.summation, max_total_synapses * sizeof( BGFLOAT ) ) );

        // create an EventQueue objet in device memory and set the pointer in device
        preSpikeQueue->createEventQueueInDevice(&allSynapses.preSpikeQueue);
}

/*
 *  Delete GPU memories.
 *
 *  @param  allSynapsesDevice  Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllSpikingSynapses::deleteSynapseDeviceStruct( void* allSynapsesDevice ) {
        AllSpikingSynapsesDeviceProperties allSynapses;

        checkCudaErrors( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

        deleteDeviceStruct( allSynapses );

        checkCudaErrors( hipFree( allSynapsesDevice ) );
}

/*
 *  Delete GPU memories.
 *  (Helper function of deleteSynapseDeviceStruct)
 *
 *  @param  allSynapsesDevice  Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 */
void AllSpikingSynapses::deleteDeviceStruct( AllSpikingSynapsesDeviceProperties& allSynapses ) {
        checkCudaErrors( hipFree( allSynapses.sourceNeuronLayoutIndex ) );
        checkCudaErrors( hipFree( allSynapses.destNeuronLayoutIndex ) );
        checkCudaErrors( hipFree( allSynapses.W ) );
        checkCudaErrors( hipFree( allSynapses.type ) );
        checkCudaErrors( hipFree( allSynapses.psr ) );
        checkCudaErrors( hipFree( allSynapses.in_use ) );
        checkCudaErrors( hipFree( allSynapses.synapse_counts ) );
        checkCudaErrors( hipFree( allSynapses.decay ) );
        checkCudaErrors( hipFree( allSynapses.tau ) );
        checkCudaErrors( hipFree( allSynapses.total_delay ) );
        checkCudaErrors( hipFree( allSynapses.summation ) );

        // delete EventQueue object in device memory.
        EventQueue::deleteEventQueueInDevice(allSynapses.preSpikeQueue);

        // Set count_neurons to 0 to avoid illegal memory deallocation 
        // at AllSpikingSynapses deconstructor.
        //allSynapses.count_neurons = 0;
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesDevice  Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllSpikingSynapses::copySynapseHostToDevice( void* allSynapsesDevice, const SimulationInfo *sim_info, const ClusterInfo *clr_info ) { // copy everything necessary
        copySynapseHostToDevice( allSynapsesDevice, clr_info->totalClusterNeurons, sim_info->maxSynapsesPerNeuron );
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesDevice     Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSpikingSynapses::copySynapseHostToDevice( void* allSynapsesDevice, int num_neurons, int maxSynapsesPerNeuron ) { // copy everything necessary
        AllSpikingSynapsesDeviceProperties allSynapses;

        checkCudaErrors( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

        copyHostToDevice( allSynapsesDevice, allSynapses, num_neurons, maxSynapsesPerNeuron );
}

/*
 *  Copy all synapses' data from host to device.
 *  (Helper function of copySynapseHostToDevice)
 *
 *  @param  allSynapsesDevice     Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSpikingSynapses::copyHostToDevice( void* allSynapsesDevice, AllSpikingSynapsesDeviceProperties& allSynapses, int num_neurons, int maxSynapsesPerNeuron ) { // copy everything necessary 
        BGSIZE max_total_synapses = maxSynapsesPerNeuron * num_neurons;

        allSynapses.maxSynapsesPerNeuron = maxSynapsesPerNeuron;
        allSynapses.total_synapse_counts = total_synapse_counts;
        allSynapses.count_neurons = count_neurons;
        checkCudaErrors( hipMemcpy ( allSynapsesDevice, &allSynapses, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyHostToDevice ) );

        // Set count_neurons to 0 to avoid illegal memory deallocation 
        // at AllSpikingSynapses deconstructor.
        allSynapses.count_neurons = 0;

        checkCudaErrors( hipMemcpy ( allSynapses.sourceNeuronLayoutIndex, sourceNeuronLayoutIndex,
                max_total_synapses * sizeof( int ), hipMemcpyHostToDevice ) );
        checkCudaErrors( hipMemcpy ( allSynapses.destNeuronLayoutIndex, destNeuronLayoutIndex,
                max_total_synapses * sizeof( int ),  hipMemcpyHostToDevice ) );
        checkCudaErrors( hipMemcpy ( allSynapses.W, W,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        checkCudaErrors( hipMemcpy ( allSynapses.type, type,
                max_total_synapses * sizeof( synapseType ), hipMemcpyHostToDevice ) );
        checkCudaErrors( hipMemcpy ( allSynapses.psr, psr,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        checkCudaErrors( hipMemcpy ( allSynapses.in_use, in_use,
                max_total_synapses * sizeof( bool ), hipMemcpyHostToDevice ) );
        checkCudaErrors( hipMemcpy ( allSynapses.synapse_counts, synapse_counts,
                        num_neurons * sizeof( BGSIZE ), hipMemcpyHostToDevice ) );
        checkCudaErrors( hipMemcpy ( allSynapses.decay, decay,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        checkCudaErrors( hipMemcpy ( allSynapses.tau, tau,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        checkCudaErrors( hipMemcpy ( allSynapses.total_delay, total_delay,
                max_total_synapses * sizeof( int ), hipMemcpyHostToDevice ) );

        // copy event queue data from host to device.
        preSpikeQueue->copyEventQueueHostToDevice(allSynapses.preSpikeQueue);
}

/*
 *  Copy all synapses' data from device to host.
 *
 *  @param  allSynapsesDevice  Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllSpikingSynapses::copySynapseDeviceToHost( void* allSynapsesDevice, const SimulationInfo *sim_info, const ClusterInfo *clr_info ) {
        // copy everything necessary
        AllSpikingSynapsesDeviceProperties allSynapses;

        checkCudaErrors( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

        copyDeviceToHost( allSynapses, sim_info, clr_info );
}

/*
 *  Copy all synapses' data from device to host.
 *  (Helper function of copySynapseDeviceToHost)
 *
 *  @param  allSynapsesDevice     Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  sim_info              SimulationInfo to refer from.
 *  @param  clr_info              ClusterInfo to refer from.
 */
void AllSpikingSynapses::copyDeviceToHost( AllSpikingSynapsesDeviceProperties& allSynapses, const SimulationInfo *sim_info, const ClusterInfo *clr_info ) {
        int num_neurons = clr_info->totalClusterNeurons;
        BGSIZE max_total_synapses = sim_info->maxSynapsesPerNeuron * num_neurons;

        checkCudaErrors( hipMemcpy ( synapse_counts, allSynapses.synapse_counts,
                num_neurons * sizeof( BGSIZE ), hipMemcpyDeviceToHost ) );
        maxSynapsesPerNeuron = allSynapses.maxSynapsesPerNeuron;
        total_synapse_counts = allSynapses.total_synapse_counts;
        count_neurons = allSynapses.count_neurons;

        // Set count_neurons to 0 to avoid illegal memory deallocation 
        // at AllSpikingSynapses deconstructor.
        allSynapses.count_neurons = 0;

        checkCudaErrors( hipMemcpy ( sourceNeuronLayoutIndex, allSynapses.sourceNeuronLayoutIndex,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( destNeuronLayoutIndex, allSynapses.destNeuronLayoutIndex,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( W, allSynapses.W,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( type, allSynapses.type,
                max_total_synapses * sizeof( synapseType ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( psr, allSynapses.psr,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( in_use, allSynapses.in_use,
                max_total_synapses * sizeof( bool ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( decay, allSynapses.decay,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( tau, allSynapses.tau,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( total_delay, allSynapses.total_delay,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );

        // copy event queue data from device to host.
        preSpikeQueue->copyEventQueueDeviceToHost(allSynapses.preSpikeQueue);
}

/*
 *  Get synapse_counts in AllSynapses struct on device memory.
 *
 *  @param  allSynapsesDevice  Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllSpikingSynapses::copyDeviceSynapseCountsToHost(void* allSynapsesDevice, const ClusterInfo *clr_info)
{
        AllSpikingSynapsesDeviceProperties allSynapses;
        int neuron_count = clr_info->totalClusterNeurons;

        checkCudaErrors( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( synapse_counts, allSynapses.synapse_counts, neuron_count * sizeof( BGSIZE ), hipMemcpyDeviceToHost ) );

        // Set count_neurons to 0 to avoid illegal memory deallocation 
        // at AllSpikingSynapses deconstructor.
        //allSynapses.count_neurons = 0;
}

/* 
 *  Get sourceNeuronLayoutIndex and in_use in AllSynapses struct on device memory.
 *
 *  @param  allSynapsesDevice  Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllSpikingSynapses::copyDeviceSourceNeuronIdxToHost(void* allSynapsesDevice, const SimulationInfo *sim_info, const ClusterInfo *clr_info)
{
        AllSpikingSynapsesDeviceProperties allSynapses;
        BGSIZE max_total_synapses = sim_info->maxSynapsesPerNeuron * clr_info->totalClusterNeurons;

        checkCudaErrors( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( sourceNeuronLayoutIndex, allSynapses.sourceNeuronLayoutIndex,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( in_use, allSynapses.in_use,
                max_total_synapses * sizeof( bool ), hipMemcpyDeviceToHost ) );
       
        // Set count_neurons to 0 to avoid illegal memory deallocation 
        // at AllSpikingSynapses deconstructor.
        //allSynapses.count_neurons = 0;
}

/*
 *  Set some parameters used for advanceSynapsesDevice.
 */
void AllSpikingSynapses::setAdvanceSynapsesDeviceParams()
{
    setSynapseClassID();
}

/**
 *  Set synapse class ID defined by enumClassSynapses for the caller's Synapse class.
 *  The class ID will be set to classSynapses_d in device memory,
 *  and the classSynapses_d will be referred to call a device function for the
 *  particular synapse class.
 *  Because we cannot use virtual function (Polymorphism) in device functions,
 *  we use this scheme.
 *  Note: we used to use a function pointer; however, it caused the growth_cuda crash
 *  (see issue#137).
 */
void AllSpikingSynapses::setSynapseClassID()
{
    enumClassSynapses classSynapses_h = classAllSpikingSynapses;

    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(classSynapses_d), &classSynapses_h, sizeof(enumClassSynapses)) );
}

/*
 * Process inter clusters outgoing spikes.
 *
 *  @param  allSynapsesDevice     Reference to the AllSpikingSynapsesDeviceProperties struct
 *                                on device memory.
 */
void AllSpikingSynapses::processInterClustesOutgoingSpikes(void* allSynapsesDevice)
{
    // copy everything necessary from device to host
    AllSpikingSynapsesDeviceProperties allSynapses;

    checkCudaErrors( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

    // process inter clusters outgoing spikes 
    preSpikeQueue->processInterClustersOutgoingEvents(allSynapses.preSpikeQueue);
}

/*
 * Process inter clusters incoming spikes.
 *
 *  @param  allSynapsesDevice     Reference to the AllSpikingSynapsesDeviceProperties struct
 *                                on device memory.
 */
void AllSpikingSynapses::processInterClustesIncomingSpikes(void* allSynapsesDevice)
{
    // copy everything necessary from host to device
    AllSpikingSynapsesDeviceProperties allSynapses;

    checkCudaErrors( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

    // process inter clusters incoming spikes 
    preSpikeQueue->processInterClustersIncomingEvents(allSynapses.preSpikeQueue);
}

/*
 *  Advance all the Synapses in the simulation.
 *  Update the state of all synapses for a time step.
 *
 *  @param  allSynapsesDevice      Reference to the AllSynapsesDeviceProperties struct 
 *                                 on device memory.
 *  @param  allNeuronsDevice       Reference to the allNeurons struct on device memory.
 *  @param  synapseIndexMapDevice  Reference to the SynapseIndexMap on device memory.
 *  @param  sim_info               SimulationInfo class to read information from.
 *  @param  clr_info               ClusterInfo to refer from.
 *  @param  iStepOffset            Offset from the current simulation step.
 */
void AllSpikingSynapses::advanceSynapses(void* allSynapsesDevice, void* allNeuronsDevice, void* synapseIndexMapDevice, const SimulationInfo *sim_info, const ClusterInfo *clr_info, int iStepOffset)
{
    if (total_synapse_counts == 0)
        return;

    // CUDA parameters
    int blocksPerGrid = ( total_synapse_counts + clr_info->threadsPerBlock - 1 ) / clr_info->threadsPerBlock;

    // Advance synapses ------------->
    //advanceSpikingSynapsesDevice <<< blocksPerGrid, clr_info->threadsPerBlock >>> ( total_synapse_counts, (SynapseIndexMap*)synapseIndexMapDevice, g_simulationStep, sim_info->deltaT, (AllSpikingSynapsesDeviceProperties*)allSynapsesDevice, iStepOffset);
    advanceDSSSynapsesDevice << < blocksPerGrid, clr_info->threadsPerBlock >> > (total_synapse_counts, (SynapseIndexMap*)synapseIndexMapDevice, g_simulationStep, sim_info->deltaT, (AllDSSynapsesDeviceProperties*)allSynapsesDevice, iStepOffset);
}

/*
 * Advances synapses spike event queue state of the cluster one simulation step.
 *
 *  @param  allSynapsesDevice      Reference to the AllSynapsesDeviceProperties struct 
 *                                 on device memory.
 *  @param  iStep                  Simulation steps to advance.
 */
void AllSpikingSynapses::advanceSpikeQueue(void* allSynapsesDevice, int iStep)
{
    advanceSpikingSynapsesEventQueueDevice <<< 1, 1 >>> ((AllSpikingSynapsesDeviceProperties*)allSynapsesDevice, iStep);
}
