/*
 * AllDSSynapses_d.cu
 *
 */

#include "AllDSSynapses.h"
#include "AllSynapsesDeviceFuncs.h"
#include <hip/hip_runtime_api.h>

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesProperties  Reference to the AllDSSynapsesProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllDSSynapses::allocSynapseDeviceStruct( void** allSynapsesProperties, const SimulationInfo *sim_info, const ClusterInfo *clr_info ) {
	allocSynapseDeviceStruct( allSynapsesProperties, clr_info->totalClusterNeurons, sim_info->maxSynapsesPerNeuron, clr_info->clusterID );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesDeviceProperties     Reference to the AllDSSynapsesProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 *  @param  clusterID             The cluster ID of the cluster.
 */
void AllDSSynapses::allocSynapseDeviceStruct( void** allSynapsesDeviceProperties, int num_neurons, int maxSynapsesPerNeuron, CLUSTER_INDEX_TYPE clusterID ) {
	AllDSSynapsesProperties allSynapsesProperties;

	allocDeviceStruct( allSynapsesProperties, num_neurons, maxSynapsesPerNeuron, clusterID );

	checkCudaErrors( hipMalloc( allSynapsesDeviceProperties, sizeof( AllDSSynapsesProperties ) ) );
	checkCudaErrors( hipMemcpy ( *allSynapsesDeviceProperties, &allSynapsesProperties, sizeof( AllDSSynapsesProperties ), hipMemcpyHostToDevice ) );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *  (Helper function of allocSynapseDeviceStruct)
 *
 *  @param  allSynapsesProperties     Reference to the AllDSSynapsesProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 *  @param  clusterID             The cluster ID of the cluster.
 */
void AllDSSynapses::allocDeviceStruct( AllDSSynapsesProperties &allSynapsesProperties, int num_neurons, int maxSynapsesPerNeuron, CLUSTER_INDEX_TYPE clusterID ) {
        AllSpikingSynapses::allocDeviceStruct( allSynapsesProperties, num_neurons, maxSynapsesPerNeuron, clusterID );

        BGSIZE max_total_synapses = maxSynapsesPerNeuron * num_neurons;

        checkCudaErrors( hipMalloc( ( void ** ) &allSynapsesProperties.lastSpike, max_total_synapses * sizeof( uint64_t ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allSynapsesProperties.r, max_total_synapses * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allSynapsesProperties.u, max_total_synapses * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allSynapsesProperties.D, max_total_synapses * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allSynapsesProperties.U, max_total_synapses * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allSynapsesProperties.F, max_total_synapses * sizeof( BGFLOAT ) ) );
}

/*
 *  Delete GPU memories.
 *
 *  @param  allSynapsesDeviceProperties  Reference to the AllDSSynapsesProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllDSSynapses::deleteSynapseDeviceStruct( void* allSynapsesDeviceProperties ) {
	AllDSSynapsesProperties allSynapsesProperties;

	checkCudaErrors( hipMemcpy ( &allSynapsesProperties, allSynapsesDeviceProperties, sizeof( AllDSSynapsesProperties ), hipMemcpyDeviceToHost ) );

	deleteDeviceStruct( allSynapsesProperties );

	checkCudaErrors( hipFree( allSynapsesDeviceProperties ) );
}

/*
 *  Delete GPU memories.
 *  (Helper function of deleteSynapseDeviceStruct)
 *
 *  @param  allSynapsesProperties  Reference to the AllDSSynapsesProperties struct 
 *                             on device memory.
 */
void AllDSSynapses::deleteDeviceStruct( AllDSSynapsesProperties& allSynapsesProperties ) {
        checkCudaErrors( hipFree( allSynapsesProperties.lastSpike ) );
	checkCudaErrors( hipFree( allSynapsesProperties.r ) );
	checkCudaErrors( hipFree( allSynapsesProperties.u ) );
	checkCudaErrors( hipFree( allSynapsesProperties.D ) );
	checkCudaErrors( hipFree( allSynapsesProperties.U ) );
	checkCudaErrors( hipFree( allSynapsesProperties.F ) );

        AllSpikingSynapses::deleteDeviceStruct( allSynapsesProperties );
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesProperties  Reference to the AllDSSynapsesProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllDSSynapses::copySynapseHostToDevice( void* allSynapsesProperties, const SimulationInfo *sim_info, const ClusterInfo *clr_info ) { // copy everything necessary
	copySynapseHostToDevice( allSynapsesProperties, clr_info->totalClusterNeurons, sim_info->maxSynapsesPerNeuron );	
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesDeviceProperties     Reference to the AllDSSynapsesProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllDSSynapses::copySynapseHostToDevice( void* allSynapsesDeviceProperties, int num_neurons, int maxSynapsesPerNeuron ) { // copy everything necessary
	AllDSSynapsesProperties allSynapsesProperties;

        checkCudaErrors( hipMemcpy ( &allSynapsesProperties, allSynapsesDeviceProperties, sizeof( AllDSSynapsesProperties ), hipMemcpyDeviceToHost ) );
	copyHostToDevice( allSynapsesDeviceProperties, allSynapsesProperties, num_neurons, maxSynapsesPerNeuron );	
}

/*
 *  Copy all synapses' data from host to device.
 *  (Helper function of copySynapseHostToDevice)
 *
 *  @param  allSynapsesDeviceProperties     Reference to the AllDSSynapsesProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllDSSynapses::copyHostToDevice( void* allSynapsesDeviceProperties, AllDSSynapsesProperties& allSynapsesProperties, int num_neurons, int maxSynapsesPerNeuron ) { // copy everything necessary 
        AllSpikingSynapses::copyHostToDevice( allSynapsesDeviceProperties, allSynapsesProperties, num_neurons, maxSynapsesPerNeuron );

        BGSIZE max_total_synapses = maxSynapsesPerNeuron * num_neurons;

        checkCudaErrors( hipMemcpy ( allSynapsesProperties.lastSpike, lastSpike,
                max_total_synapses * sizeof( uint64_t ), hipMemcpyHostToDevice ) );
        checkCudaErrors( hipMemcpy ( allSynapsesProperties.r, r,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        checkCudaErrors( hipMemcpy ( allSynapsesProperties.u, u,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        checkCudaErrors( hipMemcpy ( allSynapsesProperties.D, D,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        checkCudaErrors( hipMemcpy ( allSynapsesProperties.U, U,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        checkCudaErrors( hipMemcpy ( allSynapsesProperties.F, F,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
}

/*
 *  Copy all synapses' data from device to host.
 *
 *  @param  allSynapsesDeviceProperties  Reference to the AllDSSynapsesProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllDSSynapses::copySynapseDeviceToHost( void* allSynapsesDeviceProperties, const SimulationInfo *sim_info, const ClusterInfo *clr_info ) {
	// copy everything necessary
	AllDSSynapsesProperties allSynapsesProperties;

        checkCudaErrors( hipMemcpy ( &allSynapsesProperties, allSynapsesDeviceProperties, sizeof( AllDSSynapsesProperties ), hipMemcpyDeviceToHost ) );

	copyDeviceToHost( allSynapsesProperties, sim_info, clr_info );
}

/*
 *  Copy all synapses' data from device to host.
 *  (Helper function of copySynapseDeviceToHost)
 *
 *  @param  allSynapsesProperties     Reference to the AllDSSynapsesProperties struct 
 *                                on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllDSSynapses::copyDeviceToHost( AllDSSynapsesProperties& allSynapsesProperties, const SimulationInfo *sim_info, const ClusterInfo *clr_info ) {
        AllSpikingSynapses::copyDeviceToHost( allSynapsesProperties, sim_info, clr_info ) ;

	int num_neurons = clr_info->totalClusterNeurons;
	BGSIZE max_total_synapses = sim_info->maxSynapsesPerNeuron * num_neurons;

        checkCudaErrors( hipMemcpy ( lastSpike, allSynapsesProperties.lastSpike,
                max_total_synapses * sizeof( uint64_t ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( r, allSynapsesProperties.r,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( u, allSynapsesProperties.u,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( D, allSynapsesProperties.D,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( U, allSynapsesProperties.U,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipMemcpy ( F, allSynapsesProperties.F,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
}

/**     
 *  Set synapse class ID defined by enumClassSynapses for the caller's Synapse class.
 *  The class ID will be set to classSynapses_d in device memory,
 *  and the classSynapses_d will be referred to call a device function for the
 *  particular synapse class.
 *  Because we cannot use virtual function (Polymorphism) in device functions,
 *  we use this scheme.
 *  Note: we used to use a function pointer; however, it caused the growth_cuda crash
 *  (see issue#137).
 */
void AllDSSynapses::setSynapseClassID()
{
    enumClassSynapses classSynapses_h = classAllDSSynapses;

    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(classSynapses_d), &classSynapses_h, sizeof(enumClassSynapses)) );
}

