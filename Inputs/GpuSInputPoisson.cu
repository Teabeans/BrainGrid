#include "hip/hip_runtime.h"
/*
 *      \file GpuSInputPoisson.cu
 *
 *      \author Fumitaka Kawasaki
 *
 *      \brief A class that performs stimulus input (implementation Poisson) on GPU.
 */

#include "GpuSInputPoisson.h"
#include <hip/hip_runtime_api.h>

/*
 * constructor
 *
 * @param[in] psi       Pointer to the simulation information
 * @param[in] parms     TiXmlElement to examine.
 */
GpuSInputPoisson::GpuSInputPoisson(SimulationInfo* psi, TiXmlElement* parms) : SInputPoisson(psi, parms)
{
}

/*
 * destructor
 */
GpuSInputPoisson::~GpuSInputPoisson()
{
}

/*
 * Initialize data.
 *
 * @param[in] psi             Pointer to the simulation information.
 * @param[in] vtClrInfo       Vector of ClusterInfo.
 */
void GpuSInputPoisson::init(SimulationInfo* psi, vector<ClusterInfo *> &vtClrInfo)
{
    SInputPoisson::init(psi, vtClrInfo);

    if (m_fSInput == false)
        return;

    // allocate GPU device memory and copy values
    allocDeviceValues(psi, vtClrInfo, m_nISIs);
}

/*
 * Terminate process.
 *
 * @param[in] psi             Pointer to the simulation information.
 * @param[in] vtClrInfo       Vector of ClusterInfo.
 */
void GpuSInputPoisson::term(SimulationInfo* psi, vector<ClusterInfo *> &vtClrInfo)
{
    if (m_fSInput)
        deleteDeviceValues(vtClrInfo);

    SInputPoisson::term(psi, vtClrInfo);
}

/*
 * Process input stimulus for each time step.
 * Apply inputs on summationPoint.
 *
 * @param[in] psi             Pointer to the simulation information.
 * @param[in] pci             ClusterInfo class to read information from.
 * @param[in] iStepOffset     Offset from the current simulation step.
 */
void GpuSInputPoisson::inputStimulus(const SimulationInfo* psi, ClusterInfo *pci, int iStepOffset)
{
    if (m_fSInput == false)
        return;

    // Set device ID
    checkCudaErrors( hipSetDevice( pci->deviceId ) );

    int neuron_count = pci->totalClusterNeurons;
    int synapse_count = pci->totalClusterNeurons;

    // add input spikes to each synapse
    inputStimulusDevice <<< pci->neuronBlocksPerGrid, pci->threadsPerBlock >>> ( neuron_count, pci->nISIs_d, pci->masks_d, psi->deltaT, m_lambda, pci->devStates_d, pci->allSynapsesDeviceSInput, pci->clusterID, iStepOffset );

    // advance synapses
    //advanceSpikingSynapsesDevice <<< pci->neuronBlocksPerGrid, pci->threadsPerBlock >>> ( synapse_count, pci->synapseIndexMapDeviceSInput, g_simulationStep, psi->deltaT, (AllSpikingSynapsesDeviceProperties*)pci->allSynapsesDeviceSInput, iStepOffset);
    advanceDSSSynapsesDevice << < pci->neuronBlocksPerGrid, pci->threadsPerBlock >> > (synapse_count, pci->synapseIndexMapDeviceSInput, g_simulationStep, psi->deltaT, (AllDSSynapsesDeviceProperties*)pci->allSynapsesDeviceSInput, iStepOffset);

    // update summation point
    applyI2SummationMap <<< pci->neuronBlocksPerGrid, pci->threadsPerBlock >>> ( neuron_count, pci->pClusterSummationMap, pci->allSynapsesDeviceSInput );
    
}

/*
 * Advance input stimulus state.
 *
 * @param[in] pci             ClusterInfo class to read information from.
 * @param[in] iStep           Simulation steps to advance.
 */
void GpuSInputPoisson::advanceSInputState(const ClusterInfo *pci, int iStep)
{
    // Advances synapses pre spike event queue state of the cluster iStep simulation step
    advanceSpikingSynapsesEventQueueDevice <<< 1, 1 >>> ((AllSpikingSynapsesDeviceProperties*)pci->allSynapsesDeviceSInput, iStep);
}

/*
 * Allocate GPU device memory and copy values
 *
 * @param[in] psi        Pointer to the simulation information.
 * @param[in] vtClrInfo  Vector of ClusterInfo.
 * @param[in] nISIs      Pointer to the interval counter.
 */
void GpuSInputPoisson::allocDeviceValues(SimulationInfo* psi, vector<ClusterInfo *> &vtClrInfo, int *nISIs )
{
    // for each cluster
    for (CLUSTER_INDEX_TYPE iCluster = 0; iCluster < vtClrInfo.size(); iCluster++) 
    {
        ClusterInfo *pci = vtClrInfo[iCluster];
        int neuron_count = pci->totalClusterNeurons;

        // Set device ID
        checkCudaErrors( hipSetDevice( pci->deviceId ) );

        // Allocate GPU device memory
        BGSIZE nISIs_d_size = neuron_count * sizeof (int);   // size of shift values
        checkCudaErrors( hipMalloc ( ( void ** ) &(pci->nISIs_d), nISIs_d_size ) );

        // Copy values into device memory
        int beginIdx = pci->clusterNeuronsBegin;
        checkCudaErrors( hipMemcpy ( pci->nISIs_d, &nISIs[beginIdx], nISIs_d_size, hipMemcpyHostToDevice ) );

        // create an input synapse layer in device
        (pci->synapsesSInput)->allocSynapseDeviceStruct( (void **)&(pci->allSynapsesDeviceSInput), neuron_count, 1, pci->clusterID ); 
        (pci->synapsesSInput)->copySynapseHostToDevice( pci->allSynapsesDeviceSInput, neuron_count, 1 );

        initSynapsesDevice <<< pci->neuronBlocksPerGrid, pci->threadsPerBlock >>> ( neuron_count, pci->allSynapsesDeviceSInput, pci->pClusterSummationMap, psi->width, psi->deltaT, m_weight );

        // allocate memory for hiprand global state
        checkCudaErrors( hipMalloc ( &(pci->devStates_d), neuron_count * sizeof( hiprandState ) ) );

        // allocate memory for synapse index map and initialize it
        SynapseIndexMap synapseIndexMap;
        BGSIZE* incomingSynapseIndexMap = new BGSIZE[neuron_count];

        BGSIZE syn_i = 0;
        for (int i = 0; i < neuron_count; i++, syn_i++)
        {
            incomingSynapseIndexMap[i] = syn_i;
        }

        checkCudaErrors( hipMalloc( ( void ** ) &synapseIndexMap.incomingSynapseIndexMap, neuron_count * sizeof( BGSIZE ) ) );
        checkCudaErrors( hipMemcpy ( synapseIndexMap.incomingSynapseIndexMap, incomingSynapseIndexMap, neuron_count * sizeof( BGSIZE ), hipMemcpyHostToDevice ) ); 
        checkCudaErrors( hipMalloc( ( void ** ) &(pci->synapseIndexMapDeviceSInput), sizeof( SynapseIndexMap ) ) );
        checkCudaErrors( hipMemcpy ( pci->synapseIndexMapDeviceSInput, &synapseIndexMap, sizeof( SynapseIndexMap ), hipMemcpyHostToDevice ) );

        delete[] incomingSynapseIndexMap;

        // allocate memory for masks for stimulus input and initialize it
        checkCudaErrors( hipMalloc ( &(pci->masks_d), neuron_count * sizeof( bool ) ) );
        checkCudaErrors( hipMemcpy ( pci->masks_d, &m_masks[beginIdx], neuron_count * sizeof( bool ), hipMemcpyHostToDevice ) ); 

        // setup seeds
        setupSeeds <<< pci->neuronBlocksPerGrid, pci->threadsPerBlock >>> ( neuron_count, pci->devStates_d, time(NULL) );
    }
}

/*
 * Dellocate GPU device memory
 *
 * @param[in] vtClrInfo  Vector of ClusterInfo.
 */
void GpuSInputPoisson::deleteDeviceValues(vector<ClusterInfo *> &vtClrInfo )
{
    // for each cluster
    for (CLUSTER_INDEX_TYPE iCluster = 0; iCluster < vtClrInfo.size(); iCluster++)
    {
        ClusterInfo *pci = vtClrInfo[iCluster];

        // Set device ID
        checkCudaErrors( hipSetDevice( pci->deviceId ) );

        checkCudaErrors( hipFree( pci->nISIs_d ) );
        checkCudaErrors( hipFree( pci->devStates_d ) );
        checkCudaErrors( hipFree( pci->masks_d ) );

        (pci->synapsesSInput)->deleteSynapseDeviceStruct( pci->allSynapsesDeviceSInput );

        // deallocate memory for synapse index map
        SynapseIndexMap synapseIndexMap;
        checkCudaErrors( hipMemcpy ( &synapseIndexMap, pci->synapseIndexMapDeviceSInput, sizeof( SynapseIndexMap ), hipMemcpyDeviceToHost ) );
        checkCudaErrors( hipFree( synapseIndexMap.incomingSynapseIndexMap ) );
        checkCudaErrors( hipFree( pci->synapseIndexMapDeviceSInput ) );
    }
}

// CUDA code for -----------------------------------------------------------------------

/*
 * Device code for adding input values to the summation map.
 *
 * @param[in] nISIs_d            Pointer to the interval counter.
 * @param[in] masks_d            Pointer to the input stimulus masks.
 * @param[in] deltaT             Time step of the simulation in second.
 * @param[in] lambda             Iinverse firing rate.
 * @param[in] devStates_d        Curand global state
 * @param[in] allSynapsesDevice  Pointer to Synapse structures in device memory.
 * @param[in] clusterID          Cluster ID.
 * @param[in] iStepOffset        Offset from the current simulation step.
 */
__global__ void inputStimulusDevice( int n, int* nISIs_d, bool* masks_d, BGFLOAT deltaT, BGFLOAT lambda, hiprandState* devStates_d, AllDSSynapsesDeviceProperties* allSynapsesDevice, CLUSTER_INDEX_TYPE clusterID, int iStepOffset )
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx >= n )
        return;

    if (masks_d[idx] == false)
        return;

    BGSIZE iSyn = idx;

    int rnISIs = nISIs_d[idx];    // load the value to a register
    if (--rnISIs <= 0)
    {
        // add a spike
        allSynapsesDevice->preSpikeQueue->addAnEvent(iSyn, clusterID, iStepOffset);

        // update interval counter (exponectially distribution ISIs, Poisson)
        hiprandState localState = devStates_d[idx];

        BGFLOAT isi = -lambda * log(hiprand_uniform( &localState ));
        // delete isi within refractoriness
        while (hiprand_uniform( &localState ) <= exp(-(isi*isi)/32))
            isi = -lambda * log(hiprand_uniform( &localState ));
        // convert isi from msec to steps
        rnISIs = static_cast<int>( (isi / 1000) / deltaT + 0.5 );
        devStates_d[idx] = localState;
    }
    nISIs_d[idx] = rnISIs;
}

/*
 * CUDA code for update summation point
 *
 * @param[in] n                  Number of neurons.
 * @param[in] summationPoint_d   SummationPoint
 * @param[in] allSynapsesDevice  Pointer to Synapse structures in device memory.
 */
__global__ void applyI2SummationMap( int n, BGFLOAT* summationPoint_d, AllDSSynapsesDeviceProperties* allSynapsesDevice ) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx >= n )
            return;

    summationPoint_d[idx] += allSynapsesDevice->psr[idx];
}

/*
 * CUDA code for setup hiprand seed
 *
 * @param[in] n                  Number of neurons.
 * @param[in] devStates_d        Curand global state
 * @param[in] seed               Seed
 */
__global__ void setupSeeds( int n, hiprandState* devStates_d, unsigned long seed )
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx >= n )
            return;

    hiprand_init( seed, idx, 0, &devStates_d[idx] );
} 
