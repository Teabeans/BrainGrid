/*
 * AllIFNeurons_d.cu
 *
 */

#include "AllIFNeurons.h"
#include <hip/hip_runtime_api.h>

/*
 *  Allocate GPU memories to store all neurons' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allNeuronsDeviceProperties   Reference to the AllIFNeuronsProperties struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllIFNeurons::allocNeuronDeviceStruct( void** allNeuronsDeviceProperties, SimulationInfo *sim_info, ClusterInfo *clr_info ) {
	AllIFNeuronsProperties allNeuronsProperties;

	allocDeviceStruct( allNeuronsProperties, sim_info, clr_info );

        checkCudaErrors( hipMalloc( allNeuronsDeviceProperties, sizeof( AllIFNeuronsProperties ) ) );
        checkCudaErrors( hipMemcpy ( *allNeuronsDeviceProperties, &allNeuronsProperties, sizeof( AllIFNeuronsProperties ), hipMemcpyHostToDevice ) );
}

/*
 *  Allocate GPU memories to store all neurons' states.
 *  (Helper function of allocNeuronDeviceStruct)
 *
 *  @param  allNeuronsProperties         Reference to the AllIFNeuronsProperties struct.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllIFNeurons::allocDeviceStruct( AllIFNeuronsProperties &allNeuronsProperties, SimulationInfo *sim_info, ClusterInfo *clr_info ) {
	int count = clr_info->totalClusterNeurons;
	int max_spikes = static_cast<int> (sim_info->epochDuration * sim_info->maxFiringRate);
 
	checkCudaErrors( hipMalloc( ( void ** ) &allNeuronsProperties.C1, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeuronsProperties.C2, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeuronsProperties.Cm, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeuronsProperties.I0, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeuronsProperties.Iinject, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeuronsProperties.Inoise, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeuronsProperties.Isyn, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeuronsProperties.Rm, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeuronsProperties.Tau, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeuronsProperties.Trefract, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeuronsProperties.Vinit, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeuronsProperties.Vm, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeuronsProperties.Vreset, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeuronsProperties.Vrest, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeuronsProperties.Vthresh, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeuronsProperties.hasFired, count * sizeof( bool ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeuronsProperties.nStepsInRefr, count * sizeof( int ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeuronsProperties.spikeCount, count * sizeof( int ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeuronsProperties.spikeCountOffset, count * sizeof( int ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeuronsProperties.summation_map, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeuronsProperties.spike_history, count * sizeof( uint64_t* ) ) );
	
	uint64_t* pSpikeHistory[count];
	for (int i = 0; i < count; i++) {
		checkCudaErrors( hipMalloc( ( void ** ) &pSpikeHistory[i], max_spikes * sizeof( uint64_t ) ) );
	}
	checkCudaErrors( hipMemcpy ( allNeuronsProperties.spike_history, pSpikeHistory,
		count * sizeof( uint64_t* ), hipMemcpyHostToDevice ) );

	// get device summation point address and set it to sim info
	clr_info->pClusterSummationMap = allNeuronsProperties.summation_map;
}

/*
 *  Delete GPU memories.
 *
 *  @param  allNeuronsDeviceProperties   Reference to the AllIFNeuronsProperties struct on device memory.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllIFNeurons::deleteNeuronDeviceStruct( void* allNeuronsDeviceProperties, const ClusterInfo *clr_info ) {
	AllIFNeuronsProperties allNeuronsProperties;

	checkCudaErrors( hipMemcpy ( &allNeuronsProperties, allNeuronsDeviceProperties, sizeof( AllIFNeuronsProperties ), hipMemcpyDeviceToHost ) );

	deleteDeviceStruct( allNeuronsProperties, clr_info );

	checkCudaErrors( hipFree( allNeuronsDeviceProperties ) );
}

/*
 *  Delete GPU memories.
 *  (Helper function of deleteNeuronDeviceStruct)
 *
 *  @param  allNeuronsProperties         Reference to the AllIFNeuronsProperties struct.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllIFNeurons::deleteDeviceStruct( AllIFNeuronsProperties& allNeuronsProperties, const ClusterInfo *clr_info ) {
	int count = clr_info->totalClusterNeurons;

	uint64_t* pSpikeHistory[count];
	checkCudaErrors( hipMemcpy ( pSpikeHistory, allNeuronsProperties.spike_history,
		count * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );
	for (int i = 0; i < count; i++) {
		checkCudaErrors( hipFree( pSpikeHistory[i] ) );
	}

	checkCudaErrors( hipFree( allNeuronsProperties.C1 ) );
	checkCudaErrors( hipFree( allNeuronsProperties.C2 ) );
	checkCudaErrors( hipFree( allNeuronsProperties.Cm ) );
	checkCudaErrors( hipFree( allNeuronsProperties.I0 ) );
	checkCudaErrors( hipFree( allNeuronsProperties.Iinject ) );
	checkCudaErrors( hipFree( allNeuronsProperties.Inoise ) );
	checkCudaErrors( hipFree( allNeuronsProperties.Isyn ) );
	checkCudaErrors( hipFree( allNeuronsProperties.Rm ) );
	checkCudaErrors( hipFree( allNeuronsProperties.Tau ) );
	checkCudaErrors( hipFree( allNeuronsProperties.Trefract ) );
	checkCudaErrors( hipFree( allNeuronsProperties.Vinit ) );
	checkCudaErrors( hipFree( allNeuronsProperties.Vm ) );
	checkCudaErrors( hipFree( allNeuronsProperties.Vreset ) );
	checkCudaErrors( hipFree( allNeuronsProperties.Vrest ) );
	checkCudaErrors( hipFree( allNeuronsProperties.Vthresh ) );
	checkCudaErrors( hipFree( allNeuronsProperties.hasFired ) );
	checkCudaErrors( hipFree( allNeuronsProperties.nStepsInRefr ) );
	checkCudaErrors( hipFree( allNeuronsProperties.spikeCount ) );
	checkCudaErrors( hipFree( allNeuronsProperties.spikeCountOffset ) );
	checkCudaErrors( hipFree( allNeuronsProperties.summation_map ) );
	checkCudaErrors( hipFree( allNeuronsProperties.spike_history ) );
}

/*
 *  Copy all neurons' data from host to device.
 *
 *  @param  allNeuronsDeviceProperties   Reference to the AllIFNeuronsProperties struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllIFNeurons::copyNeuronHostToDevice( void* allNeuronsDeviceProperties, const SimulationInfo *sim_info, const ClusterInfo *clr_info ) { 
	AllIFNeuronsProperties allNeuronsProperties;

	checkCudaErrors( hipMemcpy ( &allNeuronsProperties, allNeuronsDeviceProperties, sizeof( AllIFNeuronsProperties ), hipMemcpyDeviceToHost ) );
	copyHostToDevice( allNeuronsProperties, sim_info, clr_info );
}

/*
 *  Copy all neurons' data from host to device.
 *  (Helper function of copyNeuronHostToDevice)
 *
 *  @param  allNeuronsProperties         Reference to the AllIFNeuronsProperties struct.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllIFNeurons::copyHostToDevice( AllIFNeuronsProperties& allNeuronsProperties, const SimulationInfo *sim_info, const ClusterInfo *clr_info ) { 
	int count = clr_info->totalClusterNeurons;

	checkCudaErrors( hipMemcpy ( allNeuronsProperties.C1, C1, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeuronsProperties.C2, C2, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeuronsProperties.Cm, Cm, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeuronsProperties.I0, I0, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeuronsProperties.Iinject, Iinject, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeuronsProperties.Inoise, Inoise, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeuronsProperties.Isyn, Isyn, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeuronsProperties.Rm, Rm, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeuronsProperties.Tau, Tau, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeuronsProperties.Trefract, Trefract, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeuronsProperties.Vinit, Vinit, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeuronsProperties.Vm, Vm, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeuronsProperties.Vreset, Vreset, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeuronsProperties.Vrest, Vrest, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeuronsProperties.Vthresh, Vthresh, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeuronsProperties.hasFired, hasFired, count * sizeof( bool ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeuronsProperties.nStepsInRefr, nStepsInRefr, count * sizeof( int ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeuronsProperties.spikeCount, spikeCount, count * sizeof( int ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeuronsProperties.spikeCountOffset, spikeCountOffset, count * sizeof( int ), hipMemcpyHostToDevice ) );

        int max_spikes = static_cast<int> (sim_info->epochDuration * sim_info->maxFiringRate);
        uint64_t* pSpikeHistory[count];
        checkCudaErrors( hipMemcpy ( pSpikeHistory, allNeuronsProperties.spike_history, count * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );
        for (int i = 0; i < count; i++) {
                checkCudaErrors( hipMemcpy ( pSpikeHistory[i], spike_history[i], max_spikes * sizeof( uint64_t ), hipMemcpyHostToDevice ) );
        }
}

/*
 *  Copy all neurons' data from device to host.
 *
 *  @param  allNeuronsDeviceProperties   Reference to the AllIFNeuronsProperties struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllIFNeurons::copyNeuronDeviceToHost( void* allNeuronsDeviceProperties, const SimulationInfo *sim_info, const ClusterInfo *clr_info ) {
	AllIFNeuronsProperties allNeuronsProperties;

	checkCudaErrors( hipMemcpy ( &allNeuronsProperties, allNeuronsDeviceProperties, sizeof( AllIFNeuronsProperties ), hipMemcpyDeviceToHost ) );
	copyDeviceToHost( allNeuronsProperties, sim_info, clr_info );
}

/*
 *  Copy all neurons' data from device to host.
 *  (Helper function of copyNeuronDeviceToHost)
 *
 *  @param  allNeuronsProperties         Reference to the AllIFNeuronsProperties struct.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllIFNeurons::copyDeviceToHost( AllIFNeuronsProperties& allNeuronsProperties, const SimulationInfo *sim_info, const ClusterInfo *clr_info ) {
	int count = clr_info->totalClusterNeurons;

	checkCudaErrors( hipMemcpy ( C1, allNeuronsProperties.C1, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( C2, allNeuronsProperties.C2, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( Cm, allNeuronsProperties.C1, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( I0, allNeuronsProperties.I0, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( Iinject, allNeuronsProperties.Iinject, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( Inoise, allNeuronsProperties.Inoise, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( Isyn, allNeuronsProperties.Isyn, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( Rm, allNeuronsProperties.Rm, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( Tau, allNeuronsProperties.Tau, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( Trefract, allNeuronsProperties.Trefract, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( Vinit, allNeuronsProperties.Vinit, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( Vm, allNeuronsProperties.Vm, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( Vreset, allNeuronsProperties.Vreset, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( Vrest, allNeuronsProperties.Vrest, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( Vthresh, allNeuronsProperties.Vthresh, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( hasFired, allNeuronsProperties.hasFired, count * sizeof( bool ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( nStepsInRefr, allNeuronsProperties.nStepsInRefr, count * sizeof( int ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( spikeCount, allNeuronsProperties.spikeCount, count * sizeof( int ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( spikeCountOffset, allNeuronsProperties.spikeCountOffset, count * sizeof( int ), hipMemcpyDeviceToHost ) );

        int max_spikes = static_cast<int> (sim_info->epochDuration * sim_info->maxFiringRate);
        uint64_t* pSpikeHistory[count];
        checkCudaErrors( hipMemcpy ( pSpikeHistory, allNeuronsProperties.spike_history, count * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );
        for (int i = 0; i < count; i++) {
                checkCudaErrors( hipMemcpy ( spike_history[i], pSpikeHistory[i], max_spikes * sizeof( uint64_t ), hipMemcpyDeviceToHost ) );
        }
}

/*
 *  Copy spike history data stored in device memory to host.
 *
 *  @param  allNeuronsDeviceProperties   Reference to the AllIFNeuronsProperties struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIFNeurons::copyNeuronDeviceSpikeHistoryToHost( void* allNeuronsDeviceProperties, const SimulationInfo *sim_info, const ClusterInfo *clr_info ) 
{        
        // Set device ID
        checkCudaErrors( hipSetDevice( clr_info->deviceId ) );

        AllIFNeuronsProperties allNeuronsProperties;
        checkCudaErrors( hipMemcpy ( &allNeuronsProperties, allNeuronsDeviceProperties, sizeof( AllIFNeuronsProperties ), hipMemcpyDeviceToHost ) );        
        AllSpikingNeurons::copyDeviceSpikeHistoryToHost( allNeuronsProperties, sim_info, clr_info );
}

/*
 *  Copy spike counts data stored in device memory to host.
 *
 *  @param  allNeuronsDeviceProperties   Reference to the AllIFNeuronsProperties struct on device memory.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllIFNeurons::copyNeuronDeviceSpikeCountsToHost( void* allNeuronsDeviceProperties, const ClusterInfo *clr_info )
{
        // Set device ID
        checkCudaErrors( hipSetDevice( clr_info->deviceId ) );

        AllIFNeuronsProperties allNeuronsProperties;
        checkCudaErrors( hipMemcpy ( &allNeuronsProperties, allNeuronsDeviceProperties, sizeof( AllIFNeuronsProperties ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::copyDeviceSpikeCountsToHost( allNeuronsProperties, clr_info );
}

/*
 *  Update the state of all neurons for a time step
 *  Notify outgoing synapses if neuron has fired.
 *
 *  @param  synapses               Reference to the allSynapses struct on host memory.
 *  @param  allNeuronsProperties       Reference to the AllIFNeuronsProperties struct 
 *                                 on device memory.
 *  @param  allSynapsesProperties      Reference to the allSynapsesProperties struct 
 *                                 on device memory.
 *  @param  sim_info               SimulationInfo to refer from.
 *  @param  randNoise              Reference to the random noise array.
 *  @param  synapseIndexMapDevice  Reference to the SynapseIndexMap on device memory.
 *  @param  clr_info               ClusterInfo to refer from.
 *  @param  iStepOffset            Offset from the current simulation step.
 */
void AllIFNeurons::advanceNeurons( IAllSynapses &synapses, void* allNeuronsProperties, void* allSynapsesProperties, const SimulationInfo *sim_info, float* randNoise, SynapseIndexMap* synapseIndexMapDevice, const ClusterInfo *clr_info, int iStepOffset )
{
}
