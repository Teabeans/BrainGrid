#include "AllLIFNeurons.h"
#include "AllNeuronsDeviceFuncs.h"

#include <hip/hip_runtime_api.h>

/*
 *  Update the state of all neurons for a time step
 *  Notify outgoing synapses if neuron has fired.
 *
 *  @param  synapses               Reference to the allSynapses struct on host memory.
 *  @param  allNeuronsProperties       Reference to the allNeuronsProperties struct 
 *                                 on device memory.
 *  @param  allSynapsesProperties      Reference to the allSynapsesProperties struct 
 *                                 on device memory.
 *  @param  sim_info               SimulationInfo to refer from.
 *  @param  randNoise              Reference to the random noise array.
 *  @param  synapseIndexMapDevice  Reference to the SynapseIndexMap on device memory.
 *  @param  clr_info               ClusterInfo to refer from.
 *  @param  iStepOffset            Offset from the current simulation step.
 */
void AllLIFNeurons::advanceNeurons( IAllSynapses &synapses, void* allNeuronsProperties, void* allSynapsesProperties, const SimulationInfo *sim_info, float* randNoise, SynapseIndexMap* synapseIndexMapDevice, const ClusterInfo *clr_info, int iStepOffset )
{
    DEBUG (
    int deviceId;
    checkCudaErrors( hipGetDevice( &deviceId ) );
    assert(deviceId == clr_info->deviceId);
    ); // end DEBUG

    int neuron_count = clr_info->totalClusterNeurons;
    int maxSpikes = (int)((sim_info->epochDuration * sim_info->maxFiringRate));

    // CUDA parameters
    const int threadsPerBlock = 256;
    int blocksPerGrid = ( neuron_count + threadsPerBlock - 1 ) / threadsPerBlock;

    // Advance neurons ------------->
    advanceLIFNeuronsDevice <<< blocksPerGrid, threadsPerBlock >>> ( neuron_count, sim_info->maxSynapsesPerNeuron, maxSpikes, sim_info->deltaT, g_simulationStep, randNoise, (AllIFNeuronsProperties *)allNeuronsProperties, (AllSpikingSynapsesProperties*)allSynapsesProperties, synapseIndexMapDevice, m_fAllowBackPropagation, iStepOffset );
}

