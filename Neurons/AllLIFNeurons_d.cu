#include "AllLIFNeurons.h"
#include "AllNeuronsDeviceFuncs.h"

#include <hip/hip_runtime_api.h>

/*
 *  Update the state of all neurons for a time step
 *  Notify outgoing synapses if neuron has fired.
 *
 *  @param  synapses               Reference to the allSynapses struct on host memory.
 *  @param  allNeuronsDevice       Reference to the allNeuronsDeviceProperties struct 
 *                                 on device memory.
 *  @param  allSynapsesDevice      Reference to the allSynapsesDeviceProperties struct 
 *                                 on device memory.
 *  @param  sim_info               SimulationInfo to refer from.
 *  @param  randNoise              Reference to the random noise array.
 *  @param  synapseIndexMapDevice  Reference to the SynapseIndexMap on device memory.
 *  @param  clr_info               ClusterInfo to refer from.
 *  @param  iStepOffset            Offset from the current simulation step.
 */
void AllLIFNeurons::advanceNeurons( IAllSynapses &synapses, void* allNeuronsDevice, void* allSynapsesDevice, const SimulationInfo *sim_info, float* randNoise, SynapseIndexMap* synapseIndexMapDevice, const ClusterInfo *clr_info, int iStepOffset )
{
    DEBUG (
    int deviceId;
    checkCudaErrors( hipGetDevice( &deviceId ) );
    assert(deviceId == clr_info->deviceId);
    ); // end DEBUG

    int neuron_count = clr_info->totalClusterNeurons;
    int maxSpikes = (int)((sim_info->epochDuration * sim_info->maxFiringRate));

    // Advance neurons ------------->
    advanceLIFNeuronsDevice <<< clr_info->neuronBlocksPerGrid, clr_info->threadsPerBlock >>> ( neuron_count, maxSpikes, sim_info->deltaT, g_simulationStep, randNoise, (AllIFNeuronsDeviceProperties *)allNeuronsDevice, (AllSpikingSynapsesDeviceProperties*)allSynapsesDevice, synapseIndexMapDevice, m_fAllowBackPropagation, iStepOffset );
}

