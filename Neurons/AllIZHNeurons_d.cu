/*
 * AllIZHNeurons.cu
 *
 */

#include "AllSpikingSynapses.h"
#include "AllIZHNeurons.h"
#include "AllNeuronsDeviceFuncs.h"

#include <hip/hip_runtime_api.h>

/*
 *  Allocate GPU memories to store all neurons' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allNeuronsDevice   Reference to the AllIZHNeuronsDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllIZHNeurons::allocNeuronDeviceStruct( void** allNeuronsDevice, SimulationInfo *sim_info, ClusterInfo *clr_info ) {
	AllIZHNeuronsDeviceProperties allNeurons;

	allocDeviceStruct( allNeurons, sim_info, clr_info );

        checkCudaErrors( hipMalloc( allNeuronsDevice, sizeof( AllIZHNeuronsDeviceProperties ) ) );
        checkCudaErrors( hipMemcpy ( *allNeuronsDevice, &allNeurons, sizeof( AllIZHNeuronsDeviceProperties ), hipMemcpyHostToDevice ) );
}

/*
 *  Allocate GPU memories to store all neurons' states.
 *  (Helper function of allocNeuronDeviceStruct)
 *
 *  @param  allNeurons         Reference to the AllIZHNeuronsDeviceProperties struct.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllIZHNeurons::allocDeviceStruct( AllIZHNeuronsDeviceProperties &allNeurons, SimulationInfo *sim_info, ClusterInfo *clr_info ) {
	int count = clr_info->totalClusterNeurons;

	AllIFNeurons::allocDeviceStruct( allNeurons, sim_info, clr_info );
 
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.Aconst, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.Bconst, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.Cconst, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.Dconst, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.u, count * sizeof( BGFLOAT ) ) );
	checkCudaErrors( hipMalloc( ( void ** ) &allNeurons.C3, count * sizeof( BGFLOAT ) ) );
}

/*
 *  Delete GPU memories.
 *
 *  @param  allNeuronsDevice   Reference to the AllIZHNeuronsDeviceProperties struct 
 *                             on device memory.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllIZHNeurons::deleteNeuronDeviceStruct( void* allNeuronsDevice, const ClusterInfo *clr_info ) {
	AllIZHNeuronsDeviceProperties allNeurons;

	checkCudaErrors( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );

	deleteDeviceStruct( allNeurons, clr_info );

	checkCudaErrors( hipFree( allNeuronsDevice ) );
}

/*
 *  Delete GPU memories.
 *  (Helper function of deleteNeuronDeviceStruct)
 *
 *  @param  allNeurons         Reference to the AllIZHNeuronsDeviceProperties struct.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllIZHNeurons::deleteDeviceStruct( AllIZHNeuronsDeviceProperties& allNeurons, const ClusterInfo *clr_info ) {
	checkCudaErrors( hipFree( allNeurons.Aconst ) );
	checkCudaErrors( hipFree( allNeurons.Bconst ) );
	checkCudaErrors( hipFree( allNeurons.Cconst ) );
	checkCudaErrors( hipFree( allNeurons.Dconst ) );
	checkCudaErrors( hipFree( allNeurons.u ) );
	checkCudaErrors( hipFree( allNeurons.C3 ) );

	AllIFNeurons::deleteDeviceStruct( allNeurons, clr_info );
}

/*
 *  Copy all neurons' data from host to device.
 *
 *  @param  allNeuronsDevice   Reference to the AllIZHNeuronsDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllIZHNeurons::copyNeuronHostToDevice( void* allNeuronsDevice, const SimulationInfo *sim_info, const ClusterInfo *clr_info ) { 
	AllIZHNeuronsDeviceProperties allNeurons;

	checkCudaErrors( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
	copyHostToDevice( allNeurons, sim_info, clr_info );
}

/*
 *  Copy all neurons' data from host to device.
 *  (Helper function of copyNeuronHostToDevice)
 *
 *  @param  allNeurons         Reference to the AllIZHNeuronsDeviceProperties struct.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllIZHNeurons::copyHostToDevice( AllIZHNeuronsDeviceProperties& allNeurons, const SimulationInfo *sim_info, const ClusterInfo *clr_info ) { 
	int count = clr_info->totalClusterNeurons;

	AllIFNeurons::copyHostToDevice( allNeurons, sim_info, clr_info );

	checkCudaErrors( hipMemcpy ( allNeurons.Aconst, Aconst, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeurons.Bconst, Bconst, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeurons.Cconst, Cconst, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeurons.Dconst, Dconst, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeurons.u, u, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy ( allNeurons.C3, C3, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
}

/*
 *  Copy all neurons' data from device to host.
 *
 *  @param  allNeuronsDevice   Reference to the AllIZHNeuronsDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllIZHNeurons::copyNeuronDeviceToHost( void* allNeuronsDevice, const SimulationInfo *sim_info, const ClusterInfo *clr_info ) {
	AllIZHNeuronsDeviceProperties allNeurons;

	checkCudaErrors( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
	copyDeviceToHost( allNeurons, sim_info, clr_info );
}

/*
 *  Copy all neurons' data from device to host.
 *  (Helper function of copyNeuronDeviceToHost)
 *
 *  @param  allNeurons         Reference to the AllIZHNeuronsDeviceProperties struct.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllIZHNeurons::copyDeviceToHost( AllIZHNeuronsDeviceProperties& allNeurons, const SimulationInfo *sim_info, const ClusterInfo *clr_info ) {
	int count = clr_info->totalClusterNeurons;

	AllIFNeurons::copyDeviceToHost( allNeurons, sim_info, clr_info );

	checkCudaErrors( hipMemcpy ( Aconst, allNeurons.Aconst, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( Bconst, allNeurons.Bconst, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( Cconst, allNeurons.Cconst, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( Dconst, allNeurons.Dconst, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( u, allNeurons.u, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	checkCudaErrors( hipMemcpy ( C3, allNeurons.C3, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
}

/*
 *  Copy spike history data stored in device memory to host.
 *
 *  @param  allNeuronsDevice   Reference to the AllIZHNeuronsDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllIZHNeurons::copyNeuronDeviceSpikeHistoryToHost( void* allNeuronsDevice, const SimulationInfo *sim_info, const ClusterInfo *clr_info ) {
        AllIZHNeuronsDeviceProperties allNeurons;
        checkCudaErrors( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::copyDeviceSpikeHistoryToHost( allNeurons, sim_info, clr_info );
}

/*
 *  Copy spike counts data stored in device memory to host.
 *
 *  @param  allNeuronsDevice   Reference to the AllIZHNeuronsDeviceProperties struct 
 *                             on device memory.
 *  @param  clr_info           ClusterInfo to refer from.
 */
void AllIZHNeurons::copyNeuronDeviceSpikeCountsToHost( void* allNeuronsDevice, const ClusterInfo *clr_info )
{
        AllIZHNeuronsDeviceProperties allNeurons;
        checkCudaErrors( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::copyDeviceSpikeCountsToHost( allNeurons, clr_info );
}

/**
 *  Update the state of all neurons for a time step
 *  Notify outgoing synapses if neuron has fired.
 *
 *  @param  synapses               Reference to the allSynapses struct on host memory.
 *  @param  allNeuronsDevice       Reference to the allNeurons struct on device memory.
 *  @param  allSynapsesDevice      Reference to the allSynapses struct on device memory.
 *  @param  sim_info               SimulationInfo to refer from.
 *  @param  randNoise              Reference to the random noise array.
 *  @param  synapseIndexMapDevice  Reference to the SynapseIndexMap on device memory.
 *  @param  clr_info               ClusterInfo class to read information from.
 *  @param  iStepOffset            Offset from the current simulation step.
 */
void AllIZHNeurons::advanceNeurons( IAllSynapses &synapses, void* allNeuronsDevice, void* allSynapsesDevice, const SimulationInfo *sim_info, float* randNoise, SynapseIndexMap* synapseIndexMapDevice, const ClusterInfo *clr_info, int iStepOffset)
{
    DEBUG (
    int deviceId;
    checkCudaErrors( hipGetDevice( &deviceId ) );
    assert(deviceId == clr_info->deviceId);
    ); // end DEBUG

    int neuron_count = clr_info->totalClusterNeurons;
    int maxSpikes = (int)((sim_info->epochDuration * sim_info->maxFiringRate));

    // Advance neurons ------------->
    advanceIZHNeuronsDevice <<< clr_info->blocksPerGrid, clr_info->threadsPerBlock >>> ( neuron_count, sim_info->maxSynapsesPerNeuron, maxSpikes, sim_info->deltaT, g_simulationStep, randNoise, (AllIZHNeuronsDeviceProperties *)allNeuronsDevice, (AllSpikingSynapsesDeviceProperties*)allSynapsesDevice, synapseIndexMapDevice, m_fAllowBackPropagation, iStepOffset );
}

